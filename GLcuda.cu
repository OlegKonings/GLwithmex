#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "stdio.h"
#include <hip/hip_runtime.h>
#include <math_functions.h>
#include "hip/hip_runtime.h"
#include ""

#define BLOCK_SIZE 16
#define BLOCKSIZE BLOCK_SIZE
#define AVOIDBANKCONFLICTS 0
#define USELOOPUNROLLING 1
#define TESTBLOCKS 16
#define IDC2D(i,j,ld) (((j)*(ld))+(i))

#define THREADS 64//this is 64 because for this version of ADMM group lasso, data sets will be small. For later data sets use 256
//make sure matches cpp CPPTHREADS

#define LINEAR_BLOCK_SIZE THREADS

const int blockSizeLocal=128;

//general use kernels
/////////////////////////////////////////////////////////////////////////////////////
__global__ void generateEye(float *E, const int size){
    int offset = blockIdx.x*blockDim.x + threadIdx.x;
	if(offset<(size*size)){
		int y = offset/size,x = offset - y*size;
		E[offset] = (x == y) ? 1.0f:0.0f;
	}
}
extern "C" void generateEye_wrap(float *E, const int N,const int numBlocks){
	generateEye<<<numBlocks,THREADS>>>(E,N);
}
//////////////////////////////////////////////////////////////////////////////////////

__global__ void gpu_inplace_vector_scale(float *V, const int size,const float _s){
	int offset = blockIdx.x*blockDim.x + threadIdx.x;
	if(offset<size){
		V[offset]*=_s;
	}
}
extern "C" void gpu_inplace_vector_scale_wrap(float *V, const int size,const float _s,const int numBlocks){
	gpu_inplace_vector_scale<<<numBlocks,THREADS>>>(V,size,_s);
}
//////////////////////////////////////////////////////////////////////////////////////

__global__ void gpu_vector_add(const float* __restrict__ a, const float* __restrict__ b, float* __restrict__ result,
	const int size,const bool add){
	const int offset = blockIdx.x*blockDim.x + threadIdx.x;
	const int adj= add ? 1:-1;
	if(offset<size){
		result[offset]=a[offset]+float(adj)*b[offset];
	}
	
}
extern "C" void gpu_vector_add_wrap(const float *a, const float *b, float *result, const int size, const bool add,const int numBlocks){
	gpu_vector_add<<<numBlocks,THREADS>>>(a,b,result,size,add);
}
//////////////////////////////////////////////////////////////////////////////////////

__global__ void gpu_lasso_u_update(float* __restrict__ u,const float* __restrict__ xh, const float* __restrict__ z,const int size){
	int offset = blockIdx.x*blockDim.x + threadIdx.x;
	if(offset<size){
		u[offset]+=(xh[offset]-z[offset]);
	}
}
extern "C" void gpu_lasso_u_update_wrap(float *u,const float *xh, const float *z,const int size,const int numBlocks){
	gpu_lasso_u_update<<<numBlocks,THREADS>>>(u,xh,z,size);
}
//////////////////////////////////////////////////////////////////////////////////////

__global__ void gpu_lasso_objective_helper(float* __restrict__ v0,const float* __restrict__ v1,const int size){//(v0 - v1).^2, with A
	int offset = blockIdx.x*blockDim.x + threadIdx.x;
	if(offset<size){
		float t=v0[offset]-v1[offset];
		v0[offset]=t*t;
	}
}
extern "C" void gpu_lasso_objective_helper_wrap(float *v0,const float *v1,const int size,const int numBlocks){
	gpu_lasso_objective_helper<<<numBlocks,THREADS>>>(v0,v1,size);
}
//////////////////////////////////////////////////////////////////////////////////////

__global__ void gpu_lasso_h_helper(const float* __restrict__ z, const float* __restrict__ zold,
	float* __restrict__ v_result,const float _rho,const int size){
	int offset = blockIdx.x*blockDim.x + threadIdx.x;
	if(offset<size){
		v_result[offset]= -_rho*(z[offset]-zold[offset]);
	}
}
extern "C" void gpu_lasso_h_helper_wrap(const float *z, const float *zold, float *v_result,const float _rho,const int size,const int numBlocks){
	gpu_lasso_h_helper<<<numBlocks,THREADS>>>(z,zold,v_result,_rho,size);
}
//////////////////////////////////////////////////////////////////////////////////////

__global__ void gpu_group_lasso_shrinkage(const float* __restrict__ x, float* __restrict__ z_result,
	const float kappa,const float e_norm,const int size){
	int offset = blockIdx.x*blockDim.x + threadIdx.x;
	if(offset<size){
		float temp=(1.0f-kappa/e_norm);
		z_result[offset]= (temp>0.0f) ? (temp*x[offset]):0.0f;
	}
}
extern "C" void gpu_group_lasso_shrinkage_wrap(const float *x, float *z_result,const float kappa,const float e_norm,const int size,const int numBlocks){
	gpu_group_lasso_shrinkage<<<numBlocks,THREADS>>>(x,z_result,kappa,e_norm,size);
}
//////////////////////////////////////////////////////////////////////////////////////

__global__ void d_choldc_topleft(float *M, int boffset,const int N){
    int tx = threadIdx.x,ty = threadIdx.y;

    __shared__ float topleft[BLOCK_SIZE][BLOCK_SIZE+1];
	int idx0=ty+BLOCK_SIZE*boffset,adj=tx+BLOCK_SIZE*boffset;

    topleft[ty][tx]=M[idx0*N+adj];
    __syncthreads();

    float fac;
// in this loop tx labels column, ty row
    for(int k=0;k<BLOCK_SIZE;k++){
		__syncthreads();
		fac=rsqrtf(topleft[k][k]);
		__syncthreads();
		if((ty==k)&&(tx>=k)){
			topleft[tx][ty]=(topleft[tx][ty])*fac;
		}
		__syncthreads();
		if ((ty>=tx)&&(tx>k)){
			topleft[ty][tx]=topleft[ty][tx]-topleft[tx][k]*topleft[ty][k]; 
		}
    }
    __syncthreads();
// here, tx labels column, ty row	
    if(ty>=tx){
		M[idx0*N+adj]=topleft[ty][tx];
    }
}
extern "C" void d_choldc_topleft_wrap(float *M, int boffset,const int N,const dim3 t_block){
	d_choldc_topleft<<<1,t_block>>>(M,boffset,N);
}
//////////////////////////////////////////////////////////////////////////////////////


__global__ void d_choldc_strip(float *M,int boffset,const int N){
// +1 since blockoffset labels the "topleft" position
// and boff is the working position...
    int boffx = blockIdx.x+boffset+1; 
    int tx = threadIdx.x,ty = threadIdx.y;
	int idx0=ty+BLOCK_SIZE*boffset,adj=tx+BLOCK_SIZE*boffset;
	int idx1=ty+boffx*BLOCK_SIZE,adj1=tx+boffset*BLOCK_SIZE;

    __shared__ float topleft[BLOCK_SIZE][BLOCK_SIZE+1];
    __shared__ float workingmat[BLOCK_SIZE][BLOCK_SIZE+1];

    topleft[ty][tx]=M[idx0*N+adj];
// actually read in transposed...
    workingmat[tx][ty]=M[idx1*N+adj1];

    __syncthreads();
    // now we forward-substitute for the new strip-elements...
    // one thread per column (a bit inefficient I'm afraid)
    if(ty==0){
		float dotprod;
		for(int k=0;k<BLOCK_SIZE;k++){
			dotprod=0.0f;
			for (int m=0;m<k;m++){
				dotprod+=topleft[k][m]*workingmat[m][tx];
			}
			workingmat[k][tx]=(workingmat[k][tx]-dotprod)/topleft[k][k];
		}
    }
    __syncthreads();
// is correctly transposed...
    M[idx1*N+adj1]=workingmat[tx][ty];
}
extern "C" void d_choldc_strip_wrap(float *M, int boffset,const int N,const dim3 stripgrid,const dim3 t_block){
	d_choldc_strip<<<stripgrid,t_block>>>(M,boffset,N);
}


template<int blockWork>//this gets the section norms of (x_hat_+u)
__global__ void GPU_version(const float* __restrict__ x_hat,const float* __restrict__ u, float* __restrict__ nrms, const int* __restrict__ cuml_part){

		
		__shared__ int beg,end;
		__shared__ float tot[2];
		if(threadIdx.x==0){
			beg= (blockIdx.y==0) ? 0:cuml_part[blockIdx.y-1];
			end=cuml_part[blockIdx.y];
		}
		__syncthreads();

		const int offset= threadIdx.x+blockIdx.x*blockWork;
		const int warp_idx=threadIdx.x%32;//
		//perform reduction through block
		float val=0.0f,tmp;
		int i=0,idx;
		for(;i<(blockWork>>6);i++){
			idx=(beg+offset)+i*THREADS;
			if(idx<end){
				tmp=x_hat[idx]+u[idx];
				val+=(tmp*tmp);
			}
		}
		val += __shfl(val, warp_idx + 16);
		val += __shfl(val, warp_idx + 8);
		val += __shfl(val, warp_idx + 4);
		val += __shfl(val, warp_idx + 2);
		val += __shfl(val, warp_idx + 1);
		if(threadIdx.x==0 || threadIdx.x==32){
			tot[threadIdx.x>>5]=val;
		}
		__syncthreads();
		if(threadIdx.x==0){
			atomicAdd(&nrms[blockIdx.y],(tot[0]+tot[1]));
		}
}

__global__ void lastStep(const float* __restrict__ x_hat,const float* __restrict__ u, float* __restrict__ nrms, const int* __restrict__ cuml_part,
	const int start_adr){

		__shared__ int beg,end;
		__shared__ float tot[2];
		if(threadIdx.x==0){
			beg= (blockIdx.y==0) ? 0:cuml_part[blockIdx.y-1];
			end=cuml_part[blockIdx.y];
		}
		__syncthreads();
		const int offset=beg+threadIdx.x+start_adr;
		const int warp_idx=threadIdx.x%32;//

		int i=1,adj=0,idx;
		float val=0.0f,tmp;
		for(;(offset+adj)<end;i++){
			idx=offset+adj;
			tmp=x_hat[idx]+u[idx];
			val+=(tmp*tmp);
			adj=(i<<6);
		}

		val += __shfl(val, warp_idx + 16);
		val += __shfl(val, warp_idx + 8);
		val += __shfl(val, warp_idx + 4);
		val += __shfl(val, warp_idx + 2);
		val += __shfl(val, warp_idx + 1);

		if(threadIdx.x==0 || threadIdx.x==32){
			tot[threadIdx.x>>5]=val;
		}
		__syncthreads();

		if(threadIdx.x==0){
			tmp=sqrtf((nrms[blockIdx.y]+tot[0]+tot[1]));
			nrms[blockIdx.y]=tmp;
		}
}
//now have norm (x_hat+u) in sections, need to finish shrinkage and then fill in SUM of the norm of z
//each section z will have the max((1-kappa/(norm of (x_hat(sel)+u(sel))

// will have as many blockIdx.y as K, need each value of z to be that (x_hat[idx]+u[idx])*norm(blockIdx.y), while getting the sum of all z norms in t_obj
template<int blockWork>
__global__ void adj_z_shrink(const float* __restrict__ nrms,const float* __restrict__ x_hat, const float* __restrict__ u,
	float* __restrict__ z,const int* __restrict__ cuml_part, float* __restrict__ t_obj,
	const float kappa){

		__shared__ int beg,end;
		__shared__ float tmp_pos;
		__shared__ float tot[2];
		if(threadIdx.x==0){
			beg= (blockIdx.y==0) ? 0:cuml_part[blockIdx.y-1];
			end=cuml_part[blockIdx.y];
			tmp_pos= (nrms[blockIdx.y]>0.0f) ? max(0.0f,(1.0f-kappa/nrms[blockIdx.y])):0.0f;
		}
		__syncthreads();

		const int offset= threadIdx.x+blockIdx.x*blockWork;
		const int warp_idx=threadIdx.x%32;//
		//perform reduction through block
		float val=0.0f,tmp;
		int i=0,idx;
		for(;i<(blockWork>>6);i++){
			idx=(beg+offset)+i*THREADS;
			if(idx<end){
				tmp=(x_hat[idx]+u[idx])*tmp_pos;
				val+=(tmp*tmp);
				z[idx]=tmp;
			}
		}
		val += __shfl(val, warp_idx + 16);
		val += __shfl(val, warp_idx + 8);
		val += __shfl(val, warp_idx + 4);
		val += __shfl(val, warp_idx + 2);
		val += __shfl(val, warp_idx + 1);
		if(threadIdx.x==0 || threadIdx.x==32){
			tot[threadIdx.x>>5]=val;
		}
		__syncthreads();
		if(threadIdx.x==0){
			atomicAdd(&t_obj[blockIdx.y],(tot[0]+tot[1]));	
		}		
}

__global__ void lastStep_z(const float* __restrict__ nrms,const float* __restrict__ x_hat, const float* __restrict__ u,
	float* __restrict__ z,const int* __restrict__ cuml_part, float* __restrict__ t_obj,const float kappa,
	const int start_adr,float* __restrict__ z_norm_sum){

		__shared__ int beg,end;
		__shared__ float tmp_pos;
		__shared__ float tot[2];
		if(threadIdx.x==0){
			beg= (blockIdx.y==0) ? 0:cuml_part[blockIdx.y-1];
			end=cuml_part[blockIdx.y];
			tmp_pos= (nrms[blockIdx.y]>0.0f) ? max(0.0f,(1.0f-kappa/nrms[blockIdx.y])):0.0f;
		}
		__syncthreads();

		const int offset=beg+threadIdx.x+start_adr;
		const int warp_idx=threadIdx.x%32;//

		int i=1,adj=0,idx;
		float val=0.0f,tmp;
		//fill in last new values of z and finish up getting the norm
		for(;(offset+adj)<end;i++){
			idx=offset+adj;
			tmp=(x_hat[idx]+u[idx])*tmp_pos;
			val+=(tmp*tmp);
			z[idx]=tmp;
			adj=(i<<6);
		}

		val += __shfl(val, warp_idx + 16);
		val += __shfl(val, warp_idx + 8);
		val += __shfl(val, warp_idx + 4);
		val += __shfl(val, warp_idx + 2);
		val += __shfl(val, warp_idx + 1);

		if(threadIdx.x==0 || threadIdx.x==32){
			tot[threadIdx.x>>5]=val;
		}
		__syncthreads();

		if(threadIdx.x==0){
			tmp=sqrtf((t_obj[blockIdx.y]+tot[0]+tot[1]));//this is the z_norm
			atomicAdd(&z_norm_sum[0],tmp);	
		}
}

//Note: norm_s, t_obj_arr and z_norm_sum will be memset prior to helper function call
extern "C" void z_shrinkage_wrap(float *D_z,const float *x_hat, const float *D_u,float *norm_s, float *t_obj_arr, float *z_norm_sum,
	const int adj_size,dim3 &PGrid, const int *D_cuml_part, const int rem_start, const float kappa,const int num_blx,
	hipError_t &err){

		if(adj_size==1){
			GPU_version<blockSizeLocal><<<PGrid,THREADS>>>(x_hat,D_u,norm_s,D_cuml_part);
		}else if(adj_size==2){
			GPU_version<blockSizeLocal*2><<<PGrid,THREADS>>>(x_hat,D_u,norm_s,D_cuml_part);
		}else if(adj_size==3){
			GPU_version<blockSizeLocal*4><<<PGrid,THREADS>>>(x_hat,D_u,norm_s,D_cuml_part);
		}else if(adj_size==4){
			GPU_version<blockSizeLocal*8><<<PGrid,THREADS>>>(x_hat,D_u,norm_s,D_cuml_part);
		}else{
			GPU_version<blockSizeLocal*16><<<PGrid,THREADS>>>(x_hat,D_u,norm_s,D_cuml_part);
		}

	

		PGrid.x=1;
		lastStep<<<PGrid,THREADS>>>(x_hat,D_u,norm_s,D_cuml_part,rem_start);
		
		//now have all block norms of (x_hat+u) in norm_s

		//now fill in z and get sum of z_norms for sections in z_norm_sum
		PGrid.x=num_blx;
		if(adj_size==1){
			adj_z_shrink<blockSizeLocal><<<PGrid,THREADS>>>(norm_s,x_hat,D_u,D_z,D_cuml_part,t_obj_arr,kappa);
		}else if(adj_size==2){
			adj_z_shrink<blockSizeLocal*2><<<PGrid,THREADS>>>(norm_s,x_hat,D_u,D_z,D_cuml_part,t_obj_arr,kappa);
		}else if(adj_size==3){
			adj_z_shrink<blockSizeLocal*4><<<PGrid,THREADS>>>(norm_s,x_hat,D_u,D_z,D_cuml_part,t_obj_arr,kappa);
		}else if(adj_size==4){
			adj_z_shrink<blockSizeLocal*8><<<PGrid,THREADS>>>(norm_s,x_hat,D_u,D_z,D_cuml_part,t_obj_arr,kappa);
		}else{
			adj_z_shrink<blockSizeLocal*16><<<PGrid,THREADS>>>(norm_s,x_hat,D_u,D_z,D_cuml_part,t_obj_arr,kappa);
		}
		

		PGrid.x=1;

		lastStep_z<<<PGrid,THREADS>>>(norm_s,x_hat,D_u,D_z,D_cuml_part,t_obj_arr,kappa,rem_start,z_norm_sum);
	
		//End partition loop, updated vector z by section and got the sum of the norms of Z into d_obj for later use by objective
}


